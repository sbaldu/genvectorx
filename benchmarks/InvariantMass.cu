#include "SYCLMath/VecOps.h"
#include "SYCLMath/Vector4D.h"
#include <benchmark/benchmark.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <vector>

#ifdef SINGLE_PRECISION
using arithmetic_type = float;
#else
using arithmetic_type = double;
#endif

using vec4d = ROOT::Experimental::LorentzVector<
    ROOT::Experimental::PtEtaPhiM4D<arithmetic_type>>;
template <class T> using Vector = std::vector<T>;

vec4d *GenVectors(int n) {
  vec4d *vectors = new vec4d[n];

  for (int i = 0; i < n; ++i) {
    // fill vectors
    vectors[i] = {1., 1., 1., 1.};
  }

  return vectors;
}

auto GenVectors(int n) {
  auto vectors = std::make_unique<LVector[]>(n);

  // generate n -4 momentum quantities
  std::for_each(vectors.get(), vectors.get() + n,
                [](auto &vec) -> void { vec = {1., 1., 1., 1.}; });

  return std::move(vectors);
}

void BM_InvariantMass(benchmark::State &state) {
  int count;
  hipGetDeviceCount(&count);
  hipSetDevice(count - 1);
  for (auto _ : state) {

    const auto N = state.range(0);
    size_t local_size = 128;

    vec4d *u_vectors = GenVectors(N);
    vec4d *v_vectors = GenVectors(N);

    arithmetic_type *masses = new arithmetic_type[N];

    masses = ROOT::Experimental::InvariantMasses<arithmetic_type, vec4d>(
        u_vectors, v_vectors, N, local_size);
  }
}

BENCHMARK(BM_InvariantMass)->RangeMultiplier(2)->Range(1 << 10, 1 << 20);

BENCHMARK_MAIN();
